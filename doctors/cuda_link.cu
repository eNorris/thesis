#include "cuda_link.h"

#include "quadrature.h"
#include "mesh.h"
#include "xsection.h"
#include "sourceparams.h"
#include "solverparams.h"

void reportGpuData()
{
    std::cout << "Reporting GPU resources" << std::endl;

    // Check the number of GPU resources
    int nDevices;
    hipGetDeviceCount(&nDevices);

    std::cout << "Found " << nDevices << " CUDA devices" << std::endl;

    for(unsigned int i = 0; i < nDevices; i++)
    {
        // Find a gpu
        hipDeviceProp_t props;
        checkCudaErrors(hipGetDeviceProperties(&props, i));

        std::cout << "Device " << i << ": " << props.name << " with compute "
             << props.major << "." << props.minor << " capability" << std::endl;
        std::cout << "Max threads per block: " << props.maxThreadsPerBlock << std::endl;
        std::cout << "Max grid size: " << props.maxGridSize[0] << " x " << props.maxGridSize[1] << " x " << props.maxGridSize[2] << std::endl;
        std::cout << "Memory Clock Rate (KHz): " << props.memoryClockRate << std::endl;
        std::cout << "Memory Bus Width (bits): " << props.memoryBusWidth << std::endl;
        std::cout << "Peak Memory Bandwidth (GB/s): " << (2.0*props.memoryClockRate*(props.memoryBusWidth/8)/1.0e6) << '\n' << std::endl;
    }
}

int *alloc_gpuInt(const int gpuId, const int elements, const int *data)
{
    int cudaerr;
    if((cudaerr = hipSetDevice(gpuId)) != hipSuccess)
        std::cout << "alloc_gpuInt failed to set the device with error code: " << cudaerr << std::endl;

    int *gpu_data;
    if((cudaerr = hipMalloc(&gpu_data, elements*sizeof(int))) != hipSuccess)
        std::cout << "alloc_gpuInt threw an error while allocating CUDA memory with error code: " << cudaerr << std::endl;

    if(data != NULL)
    {
        if((cudaerr = hipMemcpyAsync(gpu_data, data, elements*sizeof(int), hipMemcpyHostToDevice)) != hipSuccess)
            std::cout << "alloc_gpuInt failed while copying data with error code: " << cudaerr << std::endl;
    }

    return gpu_data;
}

float *alloc_gpuFloat(const int gpuId, const int elements, const float *cpuData)
{
    int cudaerr;
    if((cudaerr = hipSetDevice(gpuId)) != hipSuccess)
        std::cout << "alloc_gpuFloat failed to set the device with error code: " << cudaerr << std::endl;

    float *gpuData;
    if((cudaerr = hipMalloc(&gpuData, elements*sizeof(float))) != hipSuccess)
        std::cout << "alloc_gpuFloat threw an error while allocating CUDA memory with error code: " << cudaerr << std::endl;

    if(cpuData != NULL)
    {
        if((cudaerr = hipMemcpyAsync(gpuData, cpuData, elements*sizeof(float), hipMemcpyHostToDevice)) != hipSuccess)
            std::cout << "alloc_gpuFloat failed while copying data with error code: " << cudaerr << std::endl;
    }

    return gpuData;
}

void release_gpu(int gpuId, float *gpu_data)
{
    int cudaerr;
    if((cudaerr = hipSetDevice(gpuId)) != hipSuccess)
        std::cout << "release_gpu (float) failed to set the device with error code: " << cudaerr << std::endl;

    if((cudaerr = hipFree(gpu_data)) != hipSuccess)
        std::cout << "relase_gpu (float) threw an error while deallocating CUDA memory with error code: " << cudaerr << std::endl;
}

void release_gpu(int gpuId, int *gpu_data)
{
    int cudaerr;
    if((cudaerr = hipSetDevice(gpuId)) != hipSuccess)
        std::cout << "release_gpu (int) failed to set the device with error code: " << cudaerr << std::endl;

    if((cudaerr = hipFree(gpu_data)) != hipSuccess)
        std::cout << "relase_gpu (int) threw an error while deallocating int CUDA memory with error code: " << cudaerr << std::endl;
}

void updateCpuData(int gpuId, float *cpuData, float *gpuData, size_t elements, int cpuOffset)
{
    int cudaerr;
    if((cudaerr = hipSetDevice(gpuId)) != hipSuccess)
        std::cout << "updateCpuData (float) failed to set the device with error code: " << cudaerr << std::endl;

    if((cudaerr = hipMemcpyAsync(cpuData+cpuOffset, gpuData, elements*sizeof(float), hipMemcpyDeviceToHost)) != hipSuccess)
        std::cout << "updateCpuData (float) MemcpyAsync failed with error code: " << cudaerr << std::endl;
}

void updateCpuData(int gpuId, int *cpuData, int *gpuData, size_t elements, int cpuOffset)
{
    int cudaerr;
    if((cudaerr = hipSetDevice(gpuId)) != hipSuccess)
        std::cout << "updateCpuData (int) failed to set the device with error code: " << cudaerr << std::endl;

    if((cudaerr = hipMemcpyAsync(cpuData+cpuOffset, gpuData, elements*sizeof(int), hipMemcpyDeviceToHost)) != hipSuccess)
        std::cout << "updateCpuData (int) MemcpyAsync failed with error code: " << cudaerr << std::endl;
}

int launch_isoRayKernel(const Quadrature *quad, const Mesh *mesh, const XSection *xs, const SolverParams *solPar, const SourceParams *srcPar, std::vector<RAY_T> *uflux)
{

    if(uflux == NULL)
    {
        std::cout << "STOP!" << std::endl;
        return -1;
    }

    int gpuId = 0;

    // Allocate memory space for the solution vector
    float *gpuUflux = alloc_gpuFloat(gpuId, mesh->voxelCount() * xs->groupCount(), NULL);

    // Copy the xyzNode values
    float *gpuXNodes = alloc_gpuFloat(gpuId, mesh->xNodes.size(), &mesh->xNodes[0]);
    float *gpuYNodes = alloc_gpuFloat(gpuId, mesh->xNodes.size(), &mesh->yNodes[0]);
    float *gpuZNodes = alloc_gpuFloat(gpuId, mesh->xNodes.size(), &mesh->zNodes[0]);

    // Copy the dxyz values
    float *gpuDx = alloc_gpuFloat(gpuId, mesh->dx.size(), &mesh->dx[0]);
    float *gpuDy = alloc_gpuFloat(gpuId, mesh->dy.size(), &mesh->dy[0]);
    float *gpuDz = alloc_gpuFloat(gpuId, mesh->dz.size(), &mesh->dz[0]);

    // Copy the zone id number
    int *gpuZoneId = alloc_gpuInt(gpuId, mesh->zoneId.size(), &mesh->zoneId[0]);

    // Copy the atom density
    float *gpuAtomDensity = alloc_gpuFloat(gpuId, mesh->atomDensity.size(), &mesh->atomDensity[0]);

    // Copy the xs data
    float *gpuTot1d = alloc_gpuFloat(gpuId, xs->m_tot1d.size(), &xs->m_tot1d[0]);

    // Copy the source strength
    float *gpuSrcStrength = alloc_gpuFloat(gpuId, srcPar->spectraIntensity.size(), &srcPar->spectraIntensity[0]);

    //int ixSrc, iySrc, izSrc;

    unsigned int ixSrc = 0;
    unsigned int iySrc = 0;
    unsigned int izSrc = 0;

    while(mesh->xNodes[ixSrc+1] < srcPar->sourceX)
        ixSrc++;

    while(mesh->yNodes[iySrc+1] < srcPar->sourceY)
        iySrc++;

    while(mesh->zNodes[izSrc+1] < srcPar->sourceZ)
        izSrc++;

    dim3 dimGrid(mesh->xElemCt, mesh->yElemCt);
    dim3 dimBlock(mesh->zElemCt);

    std::cout << "Grid: " << dimGrid.x << "x" << dimGrid.y << ",   Block: " << dimBlock.x << "x" << dimBlock.y << std::endl;

    isoRayKernel<<<dimGrid, dimBlock>>>(
                gpuUflux,
                gpuXNodes, gpuYNodes, gpuZNodes,
                gpuDx, gpuDy, gpuDz,
                gpuZoneId,
                gpuAtomDensity,
                gpuTot1d,
                gpuSrcStrength,
                xs->groupCount(),
                mesh->xElemCt, mesh->yElemCt, mesh->zElemCt,
                srcPar->sourceX, srcPar->sourceY, srcPar->sourceZ,
                ixSrc, iySrc, izSrc);

    size_t elements = mesh->voxelCount() * xs->groupCount();
    //uflux = new RAY_T[elements];
    uflux->resize(elements);
    //hipDeviceSynchronize();

    updateCpuData(gpuId, &(*uflux)[0], gpuUflux, elements);
    //int cudaerr;
    //if((cudaerr = hipMemcpy(gpuUflux, &(*uflux)[0], elements*sizeof(float), hipMemcpyDeviceToHost)) != hipSuccess)
    //    std::cout << "launch_isoRayKernel failed while copying flux from GPU to CPU with error code "<< cudaerr << std::endl;

    release_gpu(gpuId, gpuUflux);
    release_gpu(gpuId, gpuXNodes);
    release_gpu(gpuId, gpuYNodes);
    release_gpu(gpuId, gpuZNodes);
    release_gpu(gpuId, gpuDx);
    release_gpu(gpuId, gpuDy);
    release_gpu(gpuId, gpuDz);
    release_gpu(gpuId, gpuZoneId);
    release_gpu(gpuId, gpuAtomDensity);
    release_gpu(gpuId, gpuTot1d);
    release_gpu(gpuId, gpuSrcStrength);
    //if(hipFree(gpu_data) != hipSuccess)
    //    std::cout << "alloc_gpuInt failed while copying data" << std::endl;

    return EXIT_SUCCESS;
}
