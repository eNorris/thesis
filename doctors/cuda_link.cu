#include "cuda_link.h"

#include "quadrature.h"
#include "mesh.h"
#include "xsection.h"
#include "sourceparams.h"

void reportGpuData()
{
    std::cout << "Reporting GPU resources" << std::endl;

    // Check the number of GPU resources
    int nDevices;
    hipGetDeviceCount(&nDevices);

    std::cout << "Found " << nDevices << " CUDA devices" << std::endl;

    for(unsigned int i = 0; i < nDevices; i++)
    {
        // Find a gpu
        hipDeviceProp_t props;
        checkCudaErrors(hipGetDeviceProperties(&props, i));

        std::cout << "Device " << i << ": " << props.name << " with compute "
             << props.major << "." << props.minor << " capability" << std::endl;
        std::cout << "Max threads per block: " << props.maxThreadsPerBlock << std::endl;
        std::cout << "Max grid size: " << props.maxGridSize[0] << " x " << props.maxGridSize[1] << " x " << props.maxGridSize[2] << std::endl;
        std::cout << "Memory Clock Rate (KHz): " << props.memoryClockRate << std::endl;
        std::cout << "Memory Bus Width (bits): " << props.memoryBusWidth << std::endl;
        std::cout << "Peak Memory Bandwidth (GB/s): " << (2.0*props.memoryClockRate*(props.memoryBusWidth/8)/1.0e6) << '\n' << std::endl;
    }
}

int *alloc_gpuInt(int gpuId, int elements)
{
    if(hipSetDevice(gpuId) != hipSuccess)
        std::cout << "alloc_gpu failed to set the device" << std::endl;

    //for(unsigned int i = 0; i < nDevices; i++)
    //{
    int *gpu_data;
    if(hipMalloc(&gpu_data, elements*sizeof(int)) != hipSuccess)
        std::cout << "init_gpu threw an error while allocating CUDA memory" << std::endl;
    //}

    return gpu_data;
}

float *alloc_gpuFloat(int gpuId, int elements)
{
    if(hipSetDevice(gpuId) != hipSuccess)
        std::cout << "alloc_gpu failed to set the device" << std::endl;

    //for(unsigned int i = 0; i < nDevices; i++)
    //{
    int *gpu_data;
    if(hipMalloc(&gpu_data, elements*sizeof(float)) != hipSuccess)
        std::cout << "init_gpu threw an error while allocating CUDA memory" << std::endl;
    //}

    return gpu_data;
}

void release_gpu(float *gpu_data)
{
    //int nGpu = (*gpus[0]);
    //for(int i = 0; i < nGpu; i++)
    //{
    if(hipFree(&gpu_data) != hipSuccess)
        std::cout << "relase_gpu threw an error while deallocating CUDA memory" << std::endl;
    //}
    //int **gpu_datas = new int*[nDevices+1];
    //(*gpu_datas[0]) = nDevices;
    //memcpy(gpu_datas[0], &nDevices, sizeof(int));
    //(*gpu_datas[0]) = nDevices;  // Turns nDevices into an address of a float

    //for(unsigned int i = 0; i < nDevices; i++)
    //{
    //    if(hipMalloc(&gpu_datas[i+1], elements/2*sizeof(float)) != hipSuccess)
    //        std::cout << "init_gpu threw an error while allocating CUDA memory" << std::endl;
    //}
}

void updateCpuData(float *data_cpu, float *data_gpu, size_t elements)
{
    if(hipMemcpyAsync(data_cpu, data_gpu, elements*sizeof(float), hipMemcpyDeviceToHost) != hipSuccess)
        printf("updateCpuData: Cuda Error!");
}

int launch_isoRayKernel(const Quadrature *quad, const Mesh *mesh, const XSection *xs, const std::vector<RAY_T> *uflux, const SourceParams *params)
{
    dim3 dimGrid(5);
    dim3 dimBlock(5);

    /*
    float *uflux,
    int xIndxStart, int yIndxStart, int zIndxStart,
    float *xNodes, float *yNodes, float zNodes,
    float *dx, float *dy, float *dz,
    int *zoneId,
    float *atomDensity,
    int groups,
    float *tot1d,
    flost sx, float sy, float sz,
    srcIndxX, int srcIndxY, int srcIndxZ,
    float *srcStrength
    */

    isoRayKernel<<<dimGrid, dimBlock>>>(NULL, NULL, 1, 2);
    hipDeviceSynchronize();

    return EXIT_SUCCESS;
}
