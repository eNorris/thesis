#include "hip/hip_runtime.h"
#include "cuda_link.h"

#include "quadrature.h"
#include "mesh.h"
#include "xsection.h"
#include "sourceparams.h"
#include "solverparams.h"

#include "outwriter.h"
//#include <string>
#include <stdio.h>

void reportGpuData()
{
    std::cout << "Reporting GPU resources" << std::endl;

    // Check the number of GPU resources
    int nDevices;
    hipGetDeviceCount(&nDevices);

    std::cout << "Found " << nDevices << " CUDA devices" << std::endl;

    for(unsigned int i = 0; i < nDevices; i++)
    {
        // Find a gpu
        hipDeviceProp_t props;
        checkCudaErrors(hipGetDeviceProperties(&props, i));

        std::cout << "Device " << i << ": " << props.name << " with compute "
             << props.major << "." << props.minor << " capability" << std::endl;
        std::cout << "Max threads per block: " << props.maxThreadsPerBlock << std::endl;
        std::cout << "Max grid size: " << props.maxGridSize[0] << " x " << props.maxGridSize[1] << " x " << props.maxGridSize[2] << std::endl;
        std::cout << "Memory Clock Rate (KHz): " << props.memoryClockRate << std::endl;
        std::cout << "Memory Bus Width (bits): " << props.memoryBusWidth << std::endl;
        std::cout << "Peak Memory Bandwidth (GB/s): " << (2.0*props.memoryClockRate*(props.memoryBusWidth/8)/1.0e6) << '\n' << std::endl;
    }
}

int *alloc_gpuInt(const int gpuId, const int elements, const int *data)
{
    hipError_t cudaerr;
    if((cudaerr = hipSetDevice(gpuId)) != hipSuccess)
        std::cout << "alloc_gpuInt failed to set the device with error code: " << cudaerr << ": " << hipGetErrorString(cudaerr) << std::endl;

    int *gpu_data;
    if((cudaerr = hipMalloc(&gpu_data, elements*sizeof(int))) != hipSuccess)
        std::cout << "alloc_gpuInt threw an error while allocating CUDA memory with error code: " << cudaerr << ": " << hipGetErrorString(cudaerr) << std::endl;

    if(data != NULL)
    {
        if((cudaerr = hipMemcpyAsync(gpu_data, data, elements*sizeof(int), hipMemcpyHostToDevice)) != hipSuccess)
            std::cout << "alloc_gpuInt failed while copying data with error code: " << cudaerr << ": " << hipGetErrorString(cudaerr) << std::endl;
    }

    return gpu_data;
}

float *alloc_gpuFloat(const int gpuId, const int elements, const float *cpuData)
{
    hipError_t cudaerr;
    if((cudaerr = hipSetDevice(gpuId)) != hipSuccess)
        std::cout << "alloc_gpuFloat failed to set the device with error code: " << cudaerr << ": " << hipGetErrorString(cudaerr) << std::endl;

    float *gpuData;
    if((cudaerr = hipMalloc(&gpuData, elements*sizeof(float))) != hipSuccess)
        std::cout << "alloc_gpuFloat threw an error while allocating CUDA memory with error code: " << cudaerr << ": " << hipGetErrorString(cudaerr) << std::endl;

    if(cpuData != NULL)
    {
        if((cudaerr = hipMemcpyAsync(gpuData, cpuData, elements*sizeof(float), hipMemcpyHostToDevice)) != hipSuccess)
            std::cout << "alloc_gpuFloat failed while copying data with error code: " << cudaerr << ": " << hipGetErrorString(cudaerr) << std::endl;
    }

    return gpuData;
}

void release_gpu(int gpuId, float *gpu_data)
{
    hipError_t cudaerr;
    if((cudaerr = hipSetDevice(gpuId)) != hipSuccess)
        std::cout << "release_gpu (float) failed to set the device with error code: " << cudaerr << ": " << hipGetErrorString(cudaerr) << std::endl;

    if((cudaerr = hipFree(gpu_data)) != hipSuccess)
        std::cout << "relase_gpu (float) threw an error while deallocating CUDA memory with error code: " << cudaerr << ": " << hipGetErrorString(cudaerr) << std::endl;
}

void release_gpu(int gpuId, int *gpu_data)
{
    hipError_t cudaerr;
    if((cudaerr = hipSetDevice(gpuId)) != hipSuccess)
        std::cout << "release_gpu (int) failed to set the device with error code: " << cudaerr << ": " << hipGetErrorString(cudaerr) << std::endl;

    if((cudaerr = hipFree(gpu_data)) != hipSuccess)
        std::cout << "relase_gpu (int) threw an error while deallocating int CUDA memory with error code: " << cudaerr << ": " << hipGetErrorString(cudaerr) << std::endl;
}

void updateCpuData(int gpuId, float *cpuData, float *gpuData, size_t elements, int cpuOffset)
{
    hipError_t cudaerr;
    if((cudaerr = hipSetDevice(gpuId)) != hipSuccess)
        std::cout << "updateCpuData (float) failed to set the device with error code: " << cudaerr << ": " << hipGetErrorString(cudaerr) << std::endl;

    if((cudaerr = hipMemcpyAsync(cpuData+cpuOffset, gpuData, elements*sizeof(float), hipMemcpyDeviceToHost)) != hipSuccess)
        std::cout << "updateCpuData (float) MemcpyAsync failed with error code: " << cudaerr << ": " << hipGetErrorString(cudaerr) << std::endl;
}

void updateCpuData(int gpuId, int *cpuData, int *gpuData, size_t elements, int cpuOffset)
{
    hipError_t cudaerr;
    if((cudaerr = hipSetDevice(gpuId)) != hipSuccess)
        std::cout << "updateCpuData (int) failed to set the device with error code: " << cudaerr << ": " << hipGetErrorString(cudaerr) << std::endl;

    if((cudaerr = hipMemcpyAsync(cpuData+cpuOffset, gpuData, elements*sizeof(int), hipMemcpyDeviceToHost)) != hipSuccess)
        std::cout << "updateCpuData (int) MemcpyAsync failed with error code: " << cudaerr << ": " << hipGetErrorString(cudaerr) << std::endl;
}

int launch_isoRayKernel(const Quadrature *quad, const Mesh *mesh, const XSection *xs, const SolverParams *solPar, const SourceParams *srcPar, std::vector<RAY_T> *uflux)
{
    reportGpuData();

    if(uflux == NULL)
    {
        std::cout << "STOP!" << std::endl;
        return -1;
    }

    int gpuId = 0;

    // Allocate memory space for the solution vector
    //std::cout << "Allocating uflux" << std::endl;
    float *gpuUflux = alloc_gpuFloat(gpuId, mesh->voxelCount() * xs->groupCount(), NULL);

    // Copy the xyzNode values
    float *gpuXNodes = alloc_gpuFloat(gpuId, mesh->xNodes.size(), &mesh->xNodes[0]);
    float *gpuYNodes = alloc_gpuFloat(gpuId, mesh->xNodes.size(), &mesh->yNodes[0]);
    float *gpuZNodes = alloc_gpuFloat(gpuId, mesh->xNodes.size(), &mesh->zNodes[0]);

    // Copy the dxyz values
    float *gpuDx = alloc_gpuFloat(gpuId, mesh->dx.size(), &mesh->dx[0]);
    float *gpuDy = alloc_gpuFloat(gpuId, mesh->dy.size(), &mesh->dy[0]);
    float *gpuDz = alloc_gpuFloat(gpuId, mesh->dz.size(), &mesh->dz[0]);

    // Copy the zone id number
    int *gpuZoneId = alloc_gpuInt(gpuId, mesh->zoneId.size(), &mesh->zoneId[0]);

    // Copy the atom density
    float *gpuAtomDensity = alloc_gpuFloat(gpuId, mesh->atomDensity.size(), &mesh->atomDensity[0]);

    // Copy the xs data
    float *gpuTot1d = alloc_gpuFloat(gpuId, xs->m_tot1d.size(), &xs->m_tot1d[0]);

    // Copy the source strength
    //std::cout << "Allocating source strength" << std::endl;
    float *gpuSrcStrength = alloc_gpuFloat(gpuId, srcPar->spectraIntensity.size(), &srcPar->spectraIntensity[0]);

    //int ixSrc, iySrc, izSrc;

    unsigned int ixSrc = 0;
    unsigned int iySrc = 0;
    unsigned int izSrc = 0;

    while(mesh->xNodes[ixSrc+1] < srcPar->sourceX)
        ixSrc++;

    while(mesh->yNodes[iySrc+1] < srcPar->sourceY)
        iySrc++;

    while(mesh->zNodes[izSrc+1] < srcPar->sourceZ)
        izSrc++;

    dim3 dimGrid(mesh->xElemCt, mesh->yElemCt);
    dim3 dimBlock(mesh->zElemCt);

    //std::cout << "Grid: " << dimGrid.x << "x" << dimGrid.y << ",   Block: " << dimBlock.x << "x" << dimBlock.y << std::endl;

    isoRayKernel<<<dimGrid, dimBlock>>>(
                gpuUflux,
                gpuXNodes, gpuYNodes, gpuZNodes,
                gpuDx, gpuDy, gpuDz,
                gpuZoneId,
                gpuAtomDensity,
                gpuTot1d,
                gpuSrcStrength,
                xs->groupCount(),
                mesh->xElemCt, mesh->yElemCt, mesh->zElemCt,
                srcPar->sourceX, srcPar->sourceY, srcPar->sourceZ,
                ixSrc, iySrc, izSrc);

    size_t elements = mesh->voxelCount() * xs->groupCount();
    //uflux = new RAY_T[elements];
    uflux->resize(elements);
    //hipDeviceSynchronize();

    updateCpuDataBlocking(gpuId, &(*uflux)[0], gpuUflux, elements);
    OutWriter::writeArray("uflux.dat", *uflux);


    //int cudaerr;
    //if((cudaerr = hipMemcpy(gpuUflux, &(*uflux)[0], elements*sizeof(float), hipMemcpyDeviceToHost)) != hipSuccess)
    //    std::cout << "launch_isoRayKernel failed while copying flux from GPU to CPU with error code "<< cudaerr << std::endl;

    release_gpu(gpuId, gpuUflux);
    release_gpu(gpuId, gpuXNodes);
    release_gpu(gpuId, gpuYNodes);
    release_gpu(gpuId, gpuZNodes);
    release_gpu(gpuId, gpuDx);
    release_gpu(gpuId, gpuDy);
    release_gpu(gpuId, gpuDz);
    release_gpu(gpuId, gpuZoneId);
    release_gpu(gpuId, gpuAtomDensity);
    release_gpu(gpuId, gpuTot1d);
    release_gpu(gpuId, gpuSrcStrength);

    std::cout << "Most recent CUDA Error: " << hipGetErrorString(hipGetLastError()) << std::endl;
    //if(hipFree(gpu_data) != hipSuccess)
    //    std::cout << "alloc_gpuInt failed while copying data" << std::endl;

    return EXIT_SUCCESS;
}

int launch_isoSolKernel(const Quadrature *quad, const Mesh *mesh, const XSection *xs, const SolverParams *solPar, const SourceParams *srcPar, const std::vector<RAY_T> *uFlux, std::vector<SOL_T> *scalarFlux)
{
    //std::cout << "Launching solver kernel" << std::endl;
    if(uFlux == NULL)
    {
        std::cout << "STOP!" << std::endl;
        return -1;
    }

    if(scalarFlux == NULL)
    {
        std::cout << "STOP!" << std::endl;
        return -2;
    }

    int gpuId = 0;

    std::clock_t startTime = std::clock();

    const int maxIterations = 25;
    const SOL_T epsilon = 0.01f;

    scalarFlux->resize(xs->groupCount() * mesh->voxelCount());
    std::vector<float> prevFlux(mesh->voxelCount(), 0.0f);

    std::vector<SOL_T> errMaxList;
    errMaxList.resize(xs->groupCount());

    if(uFlux == NULL && srcPar == NULL)
    {
        std::cout << "uFlux and srcPar cannot both be NULL" << std::endl;
        return 55;
    }

    // Computed the highest energy group actually used
    bool noDownscatterYet = true;
    unsigned int highestEnergy = 0;

    //std::cout << "About to do high check" << std::endl;

    while(noDownscatterYet)
    {
        SOL_T dmax = 0.0;
        unsigned int vc = mesh->voxelCount();
        for(unsigned int ir = 0; ir < vc; ir++)
        {
            dmax = (dmax > (*uFlux)[highestEnergy*vc + ir]) ? dmax : (*uFlux)[highestEnergy*vc + ir];
        }
        if(dmax <= 0.0)
        {
            std::cout << "No external source or downscatter, skipping energy group " << highestEnergy << std::endl;
            highestEnergy++;
        }
        else
        {
            noDownscatterYet = false;
        }

        if(highestEnergy >= xs->groupCount())
        {
            std::cout << "Zero flux everywhere from the raytracer" << std::endl;
            return 57;
        }
    }

    // Allocate GPU resources for the external source computation
    float *gpuUFlux = alloc_gpuFloat(gpuId, xs->groupCount()*mesh->voxelCount(), &(*uFlux)[0]);
    float *gpuColFlux = alloc_gpuFloat(gpuId, scalarFlux->size(), NULL);

    float *gpuVol = alloc_gpuFloat(gpuId, mesh->vol.size(), &mesh->vol[0]);
    float *gpuAtomDensity = alloc_gpuFloat(gpuId, mesh->atomDensity.size(), &mesh->atomDensity[0]);
    int   *gpuZoneId = alloc_gpuInt(gpuId, mesh->zoneId.size(), &mesh->zoneId[0]);

    float *gpuScatXs2d = alloc_gpuFloat(gpuId, xs->m_scat2d.size(), &xs->m_scat2d[0]);

    // Allocate additional GPU resources for the solver

    float *gpuTempFlux = alloc_gpuFloat(gpuId, mesh->voxelCount(), NULL);
    //float *gpuPreFlux = alloc_gpuFloat(gpuId, mesh->voxelCount(), NULL);
    float *gpu1stSource = alloc_gpuFloat(gpuId, mesh->voxelCount(), NULL);
    float *gpuTotalSource = alloc_gpuFloat(gpuId, mesh->voxelCount(), NULL);

    float *gpuOutboundFluxX = alloc_gpuFloat(gpuId, mesh->voxelCount(), NULL);
    float *gpuOutboundFluxY = alloc_gpuFloat(gpuId, mesh->voxelCount(), NULL);
    float *gpuOutboundFluxZ = alloc_gpuFloat(gpuId, mesh->voxelCount(), NULL);

    float *gpuAxy = alloc_gpuFloat(gpuId, mesh->Axy.size(), &mesh->Axy[0]);
    float *gpuAxz = alloc_gpuFloat(gpuId, mesh->Axz.size(), &mesh->Axz[0]);
    float *gpuAyz = alloc_gpuFloat(gpuId, mesh->Ayz.size(), &mesh->Ayz[0]);

    float *gpuMu = alloc_gpuFloat(gpuId, quad->mu.size(), &quad->mu[0]);
    float *gpuEta = alloc_gpuFloat(gpuId, quad->eta.size(), &quad->eta[0]);
    float *gpuXi = alloc_gpuFloat(gpuId, quad->zi.size(), &quad->zi[0]);
    float *gpuWt = alloc_gpuFloat(gpuId, quad->wt.size(), &quad->wt[0]);

    float *gpuTotXs1d = alloc_gpuFloat(gpuId, xs->m_tot1d.size(), &xs->m_tot1d[0]);

    // Zero the scalar flux
    int erblocks = 64;
    int ergrids = scalarFlux->size() / erblocks;
    if(scalarFlux->size() % erblocks != 0)
        ergrids += 1;  // Account for lengths not divisible by 64

    //dim3 dimGrid(mesh->xElemCt, mesh->yElemCt);
    zeroKernel<<<dim3(ergrids), dim3(erblocks)>>>(scalarFlux->size(), gpuColFlux);

    //std::cout << "Grid: " << dimGrid.x << "x" << dimGrid.y << ",   Block: " << dimBlock.x << "x" << dimBlock.y << std::endl;

    // Generate the sweep index block
    int totalSubsweeps = mesh->xElemCt + mesh->yElemCt + mesh->zElemCt - 2;
    std::vector<int> threadIndexToGlobalIndex(mesh->voxelCount());
    std::vector<int> subSweepStartIndex(totalSubsweeps);
    std::vector<int> subSweepVoxelCount(totalSubsweeps);

    // Trivial edge cases that aren't computed during the loop
    subSweepStartIndex[0] = 0;
    threadIndexToGlobalIndex[0] = 0;
    subSweepVoxelCount[totalSubsweeps-1] = 1;

    for(unsigned int iSubSweep = 0; iSubSweep < totalSubsweeps; iSubSweep++)
    {
        //std::cout << "subsweep " << iSubSweep << std::endl;

        int iSubSweepPrev = iSubSweep - 1;
        int C = (iSubSweepPrev+1) * (iSubSweepPrev+2) / 2;

        int dx = max(iSubSweepPrev+1 - (signed)mesh->xElemCt, 0);
        int dy = max(iSubSweepPrev+1 - (signed)mesh->yElemCt, 0);
        int dz = max(iSubSweepPrev+1 - (signed)mesh->zElemCt, 0);
        int dxy = max(iSubSweepPrev+1 - (signed)mesh->xElemCt - (signed)mesh->yElemCt, 0);
        int dxz = max(iSubSweepPrev+1 - (signed)mesh->xElemCt - (signed)mesh->zElemCt, 0);
        int dyz = max(iSubSweepPrev+1 - (signed)mesh->yElemCt - (signed)mesh->zElemCt, 0);

        int Lx = dx * (dx + 1) / 2;
        int Ly = dy * (dy + 1) / 2;
        int Lz = dz * (dz + 1) / 2;

        int Gxy = dxy * (dxy + 1) / 2;
        int Gxz = dxz * (dxz + 1) / 2;
        int Gyz = dyz * (dyz + 1) / 2;

        int voxPrevSubSweep = C - Lx - Ly - Lz + Gxy + Gxz + Gyz;
        subSweepStartIndex[iSubSweep] = subSweepStartIndex[iSubSweepPrev] + voxPrevSubSweep;
        subSweepVoxelCount[iSubSweepPrev] = voxPrevSubSweep;

        int voxelsSoFar = 0;
        for(int ix = 0; ix <= min(mesh->xElemCt-1, iSubSweep); ix++)
            for(int iy = 0; iy <= min(mesh->yElemCt-1, iSubSweep-ix); iy++)
            {
                int iz = iSubSweep - ix - iy;
                if(iz >= mesh->zElemCt)
                    continue;

                int ir = ix*mesh->yElemCt*mesh->zElemCt + iy*mesh->zElemCt + iz;

                //if(ix == 32 && iy == 32 && iz==8)
                //{
                //    std::cout << "ir = " << ir << std::endl;
                //}

                threadIndexToGlobalIndex[subSweepStartIndex[iSubSweep] + voxelsSoFar] = ir;
                voxelsSoFar++;
            }
    }

    int *gpuThreadIndexToGlobalIndex = alloc_gpuInt(gpuId, threadIndexToGlobalIndex.size(), &threadIndexToGlobalIndex[0]);
    //float *gpuDiffMatrix = alloc_gpuFloat(gpuId, mesh->xElemCt*mesh->yElemCt, NULL);

    dim3 dimGrid(mesh->xElemCt, mesh->yElemCt);
    dim3 dimBlock(mesh->zElemCt);

    //dim3 blockLinear(64);
    //dim3 gridLinear

    for(unsigned int ie = highestEnergy; ie < xs->groupCount(); ie++)  // for every energy group
    {
        //std::cout << "ie=" << ie << std::endl;
        int iterNum = 1;
        SOL_T maxDiff = 1.0;

        // Needs to be done before the first clearSweepKernel<<<>>> call
        int rblocks = 64;
        int rgrids = mesh->voxelCount() / rblocks;
        if(mesh->voxelCount() % rblocks != 0)
            rgrids += 1;  // Account for lengths not divisible by 64
        zeroKernel<<<dim3(rgrids), dim3(rblocks)>>>(mesh->voxelCount(), gpuTempFlux);

        // No longer needed since the src kernel is no longer an integrator
        //zeroKernelMesh<<<dimGrid, dimBlock>>>(mesh->xElemCt, mesh->yElemCt, mesh->zElemCt, gpuExtSource);

        // Compute the external source
        //std::cout << "About to launch isoSrcKernels" << std::endl;
        //for(unsigned int iSink = highestEnergy; iSink < xs->groupCount(); iSink++)
        //{
            //std::cout << "Launching iSink = " << iSink << std::endl;
        isoSrcKernel<<<dimGrid, dimBlock>>>(
                                          gpuUFlux,
                                          gpu1stSource,
                                          gpuVol, gpuAtomDensity, gpuZoneId,
                                          gpuScatXs2d,
                                          mesh->voxelCount(), xs->groupCount(), solPar->pn, highestEnergy, ie,
                                          mesh->xElemCt, mesh->yElemCt, mesh->zElemCt);
        //}

        //std::cout << "Finished src kernels" << std::endl;

        hipDeviceSynchronize();


        //std::cout << "About to write the source results" << std::endl;
        //std::vector<float> cpuExtSrc;
        //cpuExtSrc.resize(mesh->voxelCount());
        //updateCpuDataBlocking(gpuId, &cpuExtSrc[0], gpu1stSource, mesh->voxelCount());
        //char ieString[256];
        //sprintf(ieString, "%d", ie);
        //OutWriter::writeArray(std::string("gpuExtSrc") + ieString + ".dat", cpuExtSrc);
        //std::cout << "Wrote the source results" << std::endl;
        //}
        //else
        //{
        //    return 2809;
        //}

        // Zero the source array
        // No longer needed since the total is initialized with the external
        //std::cout << "Launching zero kernel" << std::endl;
        //zeroKernelMesh<<<dimGrid, dimBlock>>>(mesh->xElemCt, mesh->yElemCt, mesh->zElemCt, gpuTotalSource);

        // Calculate the down-scattering source + external source
        //std::cout << "Launching scatter kernel" << std::endl;
        downscatterKernel<<<dimGrid, dimBlock>>>(
                gpuTotalSource,
                highestEnergy, ie,
                mesh->xElemCt, mesh->yElemCt, mesh->zElemCt, xs->groupCount(), solPar->pn,
                gpuZoneId,
                gpuColFlux,
                gpuScatXs2d,
                gpuAtomDensity, gpuVol,
                gpu1stSource);

        while(iterNum <= maxIterations && maxDiff > epsilon)  // while not converged
        {
            //std::cout << "iteration: " << iterNum << std::endl;
            //clearSweepKernel<<<dimGrid, dimBlock>>>(
            //        gpuPreFlux, gpuTempFlux,
            //        mesh->xElemCt, mesh->yElemCt, mesh->zElemCt);
            clearSweepKernel<<<dimGrid, dimBlock>>>(
                    gpuColFlux, gpuTempFlux,
                    mesh->xElemCt, mesh->yElemCt, mesh->zElemCt, ie);

            for(unsigned int iang = 0; iang < quad->angleCount(); iang++)  // for every angle
            {

                //std::cout << "iang=" << iang << std::endl;
                // Find the correct direction to sweep
                //int izStart = 0;                  // Sweep start index
                int diz = 1;                      // Sweep direction
                if(quad->eta[iang] < 0)           // Condition to sweep backward
                {
                    //izStart = mesh->zElemCt - 1;  // Start at the far end
                    diz = -1;                     // Sweep toward zero
                }

                //int iyStart = 0;
                int diy = 1;
                if(quad->zi[iang] < 0)
                {
                    //iyStart = mesh->yElemCt - 1;
                    diy = -1;
                }

                //int ixStart = 0;
                int dix = 1;
                if(quad->mu[iang] < 0)
                {
                    //ixStart = mesh->xElemCt - 1;
                    dix = -1;
                }

                for(unsigned int subSweepId = 0; subSweepId < totalSubsweeps; subSweepId++)
                {

                    int raise = subSweepVoxelCount[subSweepId] % 64 == 0 ? 0 : 1;
                    dim3 dimGridS(subSweepVoxelCount[subSweepId] / 64 + raise);
                    dim3 dimBlockS(64);

                    //std::cout << "Launching the subsweep Kernel" << std::endl;

                    isoSolKernel<<<dimGridS, dimBlockS>>>(
                          gpuColFlux, gpuTempFlux,
                          gpuTotalSource,
                          gpuTotXs1d, gpuScatXs2d,
                          gpuAxy, gpuAxz, gpuAyz,
                          gpuZoneId, gpuAtomDensity, gpuVol,
                          gpuMu, gpuEta, gpuXi, gpuWt,
                          gpuOutboundFluxX, gpuOutboundFluxY, gpuOutboundFluxZ,
                          ie, iang,
                          mesh->xElemCt, mesh->yElemCt, mesh->zElemCt, xs->groupCount(), quad->angleCount(), solPar->pn,
                          dix, diy, diz,
                          subSweepStartIndex[subSweepId], subSweepVoxelCount[subSweepId], gpuThreadIndexToGlobalIndex);

                    hipDeviceSynchronize();

                    //std::cout << "Launched subSweepId=" << subSweepId <<  "(" << dimGridS.x << ", " << dimGridS.y << " : " << dimBlockS.x << ", " << dimBlockS.y << " )" << std::endl;
                    //std::cin.ignore(1024, '\n');
                    //std::cout << "Ran angle " << iang << std::endl;
                    //std::cin.get();

                }

                //std::cout << "Launched subSweepId=" << subSweepId <<  "(" << dimGridS.x << ", " << dimGridS.y << " : " << dimBlockS.x << ", " << dimBlockS.y << " )" << std::endl;
                //std::cin.ignore(1024, '\n');
                //std::cout << "Ran angle " << iang << std::endl;
                //std::cin.get();

                updateCpuDataBlocking(gpuId, &(*scalarFlux)[0], gpuTempFlux, mesh->voxelCount(), ie*mesh->voxelCount());
            } // end of all angles

            char iterString[3];  // 2 digits + NULL
            char ieString[3];  // 2 digits + NULL
            sprintf(iterString, "%d", iterNum);
            sprintf(ieString, "%d", ie);
            OutWriter::writeArray(std::string("gpuScalarFlux_") + std::string(ieString) + "_" + std::string(iterString), *scalarFlux);
            //OutWriter::writeArray(std::string("gpuScalarFlux_") + std::to_string(iterNum), *scalarFlux);

            maxDiff = -1.0e35f;
            for(unsigned int i = 0; i < mesh->voxelCount(); i++)
                maxDiff = max(((*scalarFlux)[ie*mesh->voxelCount() + i]-prevFlux[i])/(*scalarFlux)[ie*mesh->voxelCount()+i], maxDiff);

            for(unsigned int i = 0; i < mesh->voxelCount(); i++)
                prevFlux[i] = (*scalarFlux)[ie*mesh->voxelCount() + i];

            std::cout << "Max diff = " << maxDiff << std::endl;

            iterNum++;
        } // end not converged
    }  // end each energy group

    std::cout << "Time to complete: " << (std::clock() - startTime)/(double)(CLOCKS_PER_SEC/1000) << " ms" << std::endl;

    // Release the GPU resources
    release_gpu(gpuId, gpuUFlux);
    release_gpu(gpuId, gpuZoneId);
    release_gpu(gpuId, gpuAtomDensity);

    release_gpu(gpuId, gpuAxy);
    release_gpu(gpuId, gpuAxz);
    release_gpu(gpuId, gpuAyz);

    release_gpu(gpuId, gpuMu);
    release_gpu(gpuId, gpuEta);
    release_gpu(gpuId, gpuXi);
    release_gpu(gpuId, gpuWt);

    release_gpu(gpuId, gpuTotXs1d);
    release_gpu(gpuId, gpuScatXs2d);



    std::cout << "Most recent CUDA Error: " << hipGetErrorString(hipGetLastError()) << std::endl;

    return EXIT_SUCCESS;
}

/*
template <class T>
void reduce(int size, int threads, int blocks, T *d_idata, T *d_odata)
{
    int numBlocks = 0;
    int numThreads = 0;
    int maxBlocks = 64;
    int maxThreads = 256;
    //getNumBlocksAndThreads(0, size, maxBlocks, maxThreads, numBlocks, numThreads);

    hipDeviceProp_t prop;
    //int device;
   // checkCudaErrors(hipGetDevice(&gpuId));
    checkCudaErrors(hipGetDeviceProperties(&prop, gpuId));

    numThreads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
    numBlocks = (n + (numThreads * 2 - 1)) / (numThreads * 2);

    if ((float)numThreads*numBlocks > (float)prop.maxGridSize[0] * prop.maxThreadsPerBlock)
    {
        printf("n is too large, please choose a smaller number!\n");
    }

    if (numBlocks > prop.maxGridSize[0])
    {
        printf("Grid size <%d> excceeds the device capability <%d>, set block size as %d (original %d)\n",
               numBlocks, prop.maxGridSize[0], numThreads*2, numThreads);

        numBlocks /= 2;
        numThreads *= 2;
    }

    numBlocks = MIN(maxBlocks, numBlocks);

    // allocate mem for the result on host side
    T *h_odata = (T *) malloc(numBlocks*sizeof(T));

    printf("%d blocks\n\n", numBlocks);

    // allocate device memory and data
    T *d_idata = NULL;
    T *d_odata = NULL;

    checkCudaErrors(hipMalloc((void **) &d_idata, bytes));
    checkCudaErrors(hipMalloc((void **) &d_odata, numBlocks*sizeof(T)));

    // copy data directly to device memory
    checkCudaErrors(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_odata, h_idata, numBlocks*sizeof(T), hipMemcpyHostToDevice));

    // warm-up
    //reduce<T>(size, numThreads, numBlocks, whichKernel, d_idata, d_odata);

    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);

    if (isPow2(size))
    {
        switch (threads)
        {
            case 512:
                reduce6<T, 512, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 256:
                reduce6<T, 256, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 128:
                reduce6<T, 128, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 64:
                reduce6<T,  64, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 32:
                reduce6<T,  32, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 16:
                reduce6<T,  16, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case  8:
                reduce6<T,   8, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case  4:
                reduce6<T,   4, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case  2:
                reduce6<T,   2, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case  1:
                reduce6<T,   1, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;
        }
    }
    else
    {
        switch (threads)
        {
            case 512:
                reduce6<T, 512, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 256:
                reduce6<T, 256, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 128:
                reduce6<T, 128, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 64:
                reduce6<T,  64, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 32:
                reduce6<T,  32, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case 16:
                reduce6<T,  16, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case  8:
                reduce6<T,   8, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case  4:
                reduce6<T,   4, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case  2:
                reduce6<T,   2, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;

            case  1:
                reduce6<T,   1, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
                break;
        }
    }
}
*/
/*
void getNumBlocksAndThreads(int gpuId, int n, int maxBlocks, int maxThreads, int &blocks, int &threads)
{
    //get device capability, to avoid block/grid size excceed the upbound
    hipDeviceProp_t prop;
    //int device;
   // checkCudaErrors(hipGetDevice(&gpuId));
    checkCudaErrors(hipGetDeviceProperties(&prop, gpuId));

    threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
    blocks = (n + (threads * 2 - 1)) / (threads * 2);

    if ((float)threads*blocks > (float)prop.maxGridSize[0] * prop.maxThreadsPerBlock)
    {
        printf("n is too large, please choose a smaller number!\n");
    }

    if (blocks > prop.maxGridSize[0])
    {
        printf("Grid size <%d> excceeds the device capability <%d>, set block size as %d (original %d)\n",
               blocks, prop.maxGridSize[0], threads*2, threads);

        blocks /= 2;
        threads *= 2;
    }

    blocks = MIN(maxBlocks, blocks);
}
*/
